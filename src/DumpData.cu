#include "hip/hip_runtime.h"
// corresponded header file
#include "DumpData.cuh"
// necessary project headers
#include "Helper.cuh"
// 3rd party libraries headers
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#define cimg_use_tiff
#include "CImg.h"
using namespace cimg_library;
// standard libraries headers
#include <cstdio>
#include <cstring>
// system headers

namespace DumpData {

namespace {

__global__
void abs_kernel(
    hipfftReal *odata,
    const hipfftComplex *idata,
    const size_t nx, const size_t ny, const size_t nz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    int idx = iz * (nx*ny) + iy * nx + ix;
    odata[idx] = hipCabsf(idata[idx]);
}

}

namespace Device {

void real(
    std::string fname,
    const hipfftReal *d_idata,
    const size_t nx, const size_t ny, const size_t nz
) {
    fprintf(stderr,
        "[DBG] dump %ldx%ldx%ld <real> from device to \"%s\"\n",
        nx, ny, nz, fname.c_str()
    );

    CImg<float> data(nx, ny, nz);
    const size_t size = data.size() * sizeof(float);

    // pinned down the host memory region
    float *d_odata;
    cudaErrChk(hipHostRegister(data.data(), size, hipHostRegisterMapped));
    cudaErrChk(hipHostGetDevicePointer(&d_odata, data.data(), 0));

    // copy from device to host
    cudaErrChk(hipMemcpy(d_odata, d_idata, size, hipMemcpyDeviceToHost));

    // release the resources
    cudaErrChk(hipHostUnregister(data.data()));

    // save the result to file
    data.save_tiff(fname.c_str());
}

void complex(
    std::string fname,
    const hipfftComplex *d_idata,
    const size_t nx, const size_t ny, const size_t nz
) {
    fprintf(stderr,
        "[DBG] dump %ldx%ldx%ld <complex> from device to \"%s\"\n",
        nx, ny, nz, fname.c_str()
    );

    CImg<float> data(nx, ny, nz);
    const size_t size = data.size() * sizeof(float);

    // pinned down the host memory region
    float *d_odata;
    cudaErrChk(hipHostRegister(data.data(), size, hipHostRegisterMapped));
    cudaErrChk(hipHostGetDevicePointer(&d_odata, data.data(), 0));

    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(nx, nthreads.x), DIVUP(ny, nthreads.y), DIVUP(nz, nthreads.z)
    );
    abs_kernel<<<nblocks, nthreads>>>(
        d_odata,
        d_idata,
        nx, ny, nz
    );
    cudaErrChk(hipPeekAtLastError());

    // release the resources
    cudaErrChk(hipHostUnregister(data.data()));

    // save the result to file
    data.save_tiff(fname.c_str());
}

}

namespace Host {

void real(
    std::string fname,
    hipfftReal *h_idata,
    const size_t nx, const size_t ny, const size_t nz
) {
    fprintf(stderr,
        "[DBG] dump %ldx%ldx%ld <real> from host to \"%s\"\n",
        nx, ny, nz, fname.c_str()
    );

    CImg<float> data(nx, ny, nz);
    const size_t size = data.size() * sizeof(float);

    // copy to image data region
    std::memcpy(data.data(), h_idata, size);

    // save the result to file
    data.save_tiff(fname.c_str());
}

}

}
