#include "hip/hip_runtime.h"
// corresponded header file
// necessary project headers
#include "DeconvRLImpl.cuh"
#include "Helper.cuh"
#include "DumpData.cuh"
// 3rd party libraries headers
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hipfft/hipfft.h>

#define cimg_use_tiff
#include "CImg.h"
using namespace cimg_library;
// standard libraries headers
#include <cstdint>
#include <exception>
// system headers

namespace DeconvRL {

namespace PSF {

namespace {

hipArray_t psfRes = nullptr;
texture<float, hipTextureType3D, hipReadModeElementType> psfTexRef;

struct SubConstant
    : public thrust::unary_function<float, float> {
    SubConstant(const float c_)
        : c(c_) {
    }

    __host__ __device__
    float operator()(const float &p) const {
        float o = p-c;
        return (o < 0) ? 0 : o;
    }

private:
    const float c;
};

__global__
void createGrid_kernel(
    int3 *d_grid,
    const size_t nx, const size_t ny, const size_t nz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    int idx = iz * (nx*ny) + iy * nx + ix;
    d_grid[idx] = make_int3(ix, iy, iz);
}

struct MultiplyWeighting
    : public thrust::unary_function<int3, float4> {
    MultiplyWeighting(
        const float *data,
        const size_t nx_, const size_t ny_, const size_t nz_
    )
        : d_weight(data), nx(nx_), ny(ny_), nz(nz_) {
    }

    __host__ __device__
    float4 operator()(const int3 &p) const {
        const int idx = p.z * (nx*ny) + p.y * nx + p.x;
        const float w = d_weight[idx];
        return make_float4(p.x*w, p.y*w, p.z*w, w);
    }

private:
    const float *d_weight;
    size_t nx, ny, nz;
};

__global__
void simpleClone_kenel(
    float *d_tmp,
    const size_t nx, const size_t ny, const size_t nz,
    const float *d_psf,
    const size_t npx, const size_t npy, const size_t npz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    int oidx = iz * (nx*ny) + iy * nx + ix;
    // rest of the space is filled with zero
    if (ix >= npx or iy >= npy or iz >= npz) {
        d_tmp[oidx] = 0.0f;
    } else {
        int iidx = iz * (npx*npy) + iy * npx + ix;
        d_tmp[oidx] = d_psf[iidx];
    }
}

__global__
void alignCenter_kernel(
    float *odata,
    const size_t nx, const size_t ny, const size_t nz,
    const float ox, const float oy, const float oz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    // normalized coordinate
    float fx = (ix+ox+0.5f) / nx;
    float fy = (iy+oy+0.5f) / ny;
    float fz = (iz+oz+0.5f) / nz;

    // sampling from the texture
    // (coordinates are backtracked to the deviated ones)
    int idx = iz * (nx*ny) + iy * nx + ix;
    odata[idx] = tex3D(psfTexRef, fx, fy, fz);
}

__global__
void fftshift3_kernel(
    float *odata,
    const float *idata,
    const size_t nx, const size_t ny, const size_t nz
) {
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    int iz = blockIdx.z*blockDim.z + threadIdx.z;

    // skip out-of-bound threads
    if (ix >= nx or iy >= ny or iz >= nz) {
        return;
    }

    // input linear index
    const int iidx = iz * (nx*ny) + iy * nx + ix;

    // calculate wrap-around (x, y, z)
    // ... advance half cycle
    ix += nx/2, iy += ny/2, iz += nz/2;
    // ... wrap-around
    ix %= nx, iy %= ny, iz %= nz;
    // output linear index
    const int oidx = iz * (nx*ny) + iy * nx + ix;

    odata[oidx] = idata[iidx];
}

}

PSF::PSF(
    float *h_psf,
    const size_t npx_, const size_t npy_, const size_t npz_
) : npx(npx_), npy(npy_), npz(npz_) {
    nelem = npx * npy * npz;

    // create PSF memory space on device
    const size_t size = nelem * sizeof(float);
    cudaErrChk(hipMalloc(&d_psf, size));
    // copy from host to device
    cudaErrChk(hipMemcpy(d_psf, h_psf, size, hipMemcpyHostToDevice));
}

PSF::~PSF() {
    cudaErrChk(hipFree(d_psf));
}

void PSF::alignCenter(const size_t nx, const size_t ny, const size_t nz) {
    float3 centroid = findCentroid();
    fprintf(
        stderr,
        "[INF] centroid = (%.2f, %.2f, %.2f)\n",
        centroid.x, centroid.y, centroid.z
    );

    // simple clone is performed, so we can reuse the centroid location
    if (nx != npx or ny != npy or nz != npz) {
        padPSF(nx, ny, nz);
    }

    /*
     * Bind the data source to the texture.
     */
    // create hipArray for the texture.
    hipChannelFormatDesc desc = hipCreateChannelDesc(
        32, 0, 0, 0, hipChannelFormatKindFloat
    );
    hipExtent extent = make_hipExtent(npx, npy, npz);
    cudaErrChk(hipMalloc3DArray(
        &psfRes,
        &desc,      // pixel channel description
        extent,     // array dimension
        hipArrayDefault
    ));

    // copy the data to hipArray_t
    hipMemcpy3DParms parms = {0};
    parms.srcPtr = make_hipPitchedPtr(d_psf, npx * sizeof(float), npx, npy);
    parms.dstArray = psfRes;
    parms.extent = extent;
    parms.kind = hipMemcpyDeviceToDevice;
    cudaErrChk(hipMemcpy3D(&parms));

    // reconfigure the texture
    psfTexRef.normalized = true;
    // sampled data is interpolated
    psfTexRef.filterMode = hipFilterModeLinear;
    // wrap around the texture if exceeds border limit
    psfTexRef.addressMode[0] = hipAddressModeWrap;
    psfTexRef.addressMode[1] = hipAddressModeWrap;
    psfTexRef.addressMode[2] = hipAddressModeWrap;

    // start the binding
    cudaErrChk(hipBindTextureToArray(psfTexRef, psfRes));

    /*
     * Execute the alignment kernel.
     */
    // coordinate of the center of the volume
    const float3 center = make_float3(
        (npx-1)/2.0f, (npy-1)/2.0f, (npz-1)/2.0f
    );
    // offset
    const float3 offset = centroid - center;
    fprintf(stderr, "[DBG] offset = (%.2f, %.2f, %.2f)\n", offset.x, offset.y, offset.z);

    // begin resample the kernel
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(npx, nthreads.x), DIVUP(npy, nthreads.y), DIVUP(npz, nthreads.z)
    );
    alignCenter_kernel<<<nblocks, nthreads>>>(
        d_psf,
        npx, npy, npz,
        offset.x, offset.y, offset.z
    );
    cudaErrChk(hipPeekAtLastError());

    /*
     * Release the resources.
     */
    cudaErrChk(hipUnbindTexture(psfTexRef));
    cudaErrChk(hipFreeArray(psfRes));

    DumpData::Device::real("psf_aligned.tif", d_psf, npx, npy, npz);
}

void PSF::createOTF(hipfftComplex *d_otf) {
    /*
     * Prepare FFT environment.
     */
    hipfftHandle otfHdl;
    cudaErrChk(hipfftPlan3d(&otfHdl, npz, npy, npx, HIPFFT_R2C));
    // estimate resource requirements
    size_t size;
    cudaErrChk(hipfftGetSize3d(otfHdl, npz, npy, npx, HIPFFT_R2C, &size));
    fprintf(stderr, "[DBG] require %ld bytes to generate an OTF\n", size);

    /*
     * Execute the conversion.
     */
    // temporary memory space to hold the shifted PSF
    float *d_tmp;
    cudaErrChk(hipMalloc(&d_tmp, nelem * sizeof(float)));
    cudaErrChk(hipMemcpy(d_tmp, d_psf, nelem * sizeof(float), hipMemcpyDeviceToDevice));

    // fftshift
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(npx, nthreads.x), DIVUP(npy, nthreads.y), DIVUP(npz, nthreads.z)
    );
    fftshift3_kernel<<<nblocks, nthreads>>>(d_tmp, d_psf, npx, npy, npz);

    cudaErrChk(hipfftExecR2C(otfHdl, d_tmp, d_otf));

    // release FFT resource
    cudaErrChk(hipFree(d_tmp));
    cudaErrChk(hipfftDestroy(otfHdl));

    DumpData::Device::complex("otf_dump.tif", d_otf, npx/2+1, npy, npz);
}

// center the PSF to its potential centroid
float3 PSF::findCentroid() {
    /*
     * Create temporary PSF to find the centroid.
     */
    float *d_tmp;
    const size_t size = nelem * sizeof(float);
    cudaErrChk(hipMalloc(&d_tmp, size));
    // copy the raw PSF to temporary PSF
    cudaErrChk(hipMemcpy(d_tmp, d_psf, size, hipMemcpyDeviceToDevice));

    // background value is clamped in [0, +inf)
    const float bkgVal = estimateBackground();
    fprintf(stderr, "[INF] PSF background value is %.2f\n", bkgVal);
    thrust::transform(
        thrust::device,
        d_tmp, d_tmp+nelem,
        d_tmp,
        SubConstant(bkgVal)
    );

    /*
     * Generate 3-D grid for weighting.
     */
    int3 *d_grid;
    cudaErrChk(hipMalloc(&d_grid, nelem * sizeof(int3)));
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(npx, nthreads.x), DIVUP(npy, nthreads.y), DIVUP(npz, nthreads.z)
    );
    createGrid_kernel<<<nblocks, nthreads>>>(d_grid, npx, npy, npz);
    cudaErrChk(hipPeekAtLastError());

    /*
     * Calculate the centroid along weighted grid points using cleaned PSF.
     */
    float4 result = thrust::transform_reduce(
        thrust::device,
        d_grid, d_grid+nelem,
        MultiplyWeighting(d_tmp, npx, npy, npz),
        make_float4(0),
        thrust::plus<float4>()
    );

    float3 centroid = make_float3(
        result.x/result.w, result.y/result.w, result.z/result.w
    );

    // free the weight computation resources
    cudaErrChk(hipFree(d_grid));
    cudaErrChk(hipFree(d_tmp));

    return centroid;
}

float PSF::estimateBackground() {
    float sum = thrust::reduce(
        thrust::device,
        d_psf, d_psf+nelem,
        0,
        thrust::plus<float>()
    );
    return sum/nelem;
}

void PSF::padPSF(const size_t nx, const size_t ny, const size_t nz) {
    // verify the size requirement
    if (nx < npx or ny < npy or nz < npz) {
        throw std::range_error(
            "volume has to be greater or equal than the original PSF"
        );
    }

    // create new PSF memory space with the larger size
    float *d_tmp;
    cudaErrChk(hipMalloc(
        &d_tmp,
        nx * ny * nz * sizeof(float)
    ));

    // copy the original PSF to the larger space at (0, 0)
    dim3 nthreads(16, 16, 4);
    dim3 nblocks(
        DIVUP(nx, nthreads.x), DIVUP(ny, nthreads.y), DIVUP(nz, nthreads.z)
    );
    simpleClone_kenel<<<nblocks, nthreads>>>(
        d_tmp,
        nx, ny, nz,
        d_psf,
        npx, npy, npz
    );
    // swap the pointer and free the smaller space
    std::swap(d_tmp, d_psf);
    cudaErrChk(hipFree(d_tmp));

    // update PSF size
    npx = nx;
    npy = ny;
    npz = nz;
    nelem = npx * npy * npz;
    fprintf(stderr, "[DBG] update PSF size is %ldx%ldx%ld\n", npx, npy, npz);
}

}

}
